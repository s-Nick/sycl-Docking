#include "hip/hip_runtime.h"
#include "rotation.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include <stdio.h>
#include "atom.cuh"
#include <iostream>
#include "hip/hip_math_constants.h"

#define NUM_OF_BLOCKS 360

using namespace std;

inline hipError_t checkCuda(hipError_t result, int line)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s at line %d\n", hipGetErrorString(result),line);
  }
  return result;
}

/**
 * Initialize the first row of the rotation matrix as explained in the report
 */
__device__ void initFirstRow_v2(double rotation_matrix[3][3], double4 & unit_quat){
        
    rotation_matrix[0][0] =  1-2*(pow(unit_quat.y,2) + pow(unit_quat.z,2));
    
    rotation_matrix[0][1] = 2*(unit_quat.x*unit_quat.y - unit_quat.z*unit_quat.w);

    rotation_matrix[0][2] = 2*(unit_quat.x*unit_quat.z + unit_quat.y*unit_quat.w);
    for(int i= 0;i < 3;i++){
        if(rotation_matrix[0][i] == -0) rotation_matrix[0][i] = 0;
    }
    //printf("row %d values : %lf %lf %lf\n", tid,row_start[0],row_start[1],row_start[2]);


}

/**
 * Initialize the second row of the rotation matrix as explained in the report
 */

__device__ void initSecondRow_v2(double rotation_matrix[3][3], double4 & unit_quat){
    
    
    rotation_matrix[1][0] =  2*(unit_quat.x*unit_quat.y + unit_quat.z*unit_quat.w);

    rotation_matrix[1][1] = 1-2*(unit_quat.x*unit_quat.x + pow(unit_quat.z,2));//unit_quat.z*unit_quat.z);

    rotation_matrix[1][2] = 2*(unit_quat.y*unit_quat.z - unit_quat.x*unit_quat.w);
    for(int i= 0;i < 3;i++){
        if(rotation_matrix[1][i] == -0) rotation_matrix[1][i] = 0;
    }

    //printf("row %d values : %lf %lf %lf\n", tid,row_start[0],row_start[1],row_start[2]);

}

/**
 * Initialize the third row of the rotation matrix as explained in the report
 */
__device__ void initThirdRow_v2(double rotation_matrix[3][3], double4 & unit_quat){
    
    
    rotation_matrix[2][0] =  2*(unit_quat.x*unit_quat.z - unit_quat.y*unit_quat.w);

    rotation_matrix[2][1] = 2*(unit_quat.y*unit_quat.z + unit_quat.x*unit_quat.w);

    rotation_matrix[2][2] = 1-2*(pow(unit_quat.x,2) + pow(unit_quat.y,2));
    for(int i= 0;i < 3;i++){
        if(rotation_matrix[2][i] == -0) rotation_matrix[2][i] = 0;
    }
    //printf("row %d values : %lf %lf %lf\n", tid,row_start[0],row_start[1],row_start[2]);

}


/**
 * Compute the rotation as a matrix-vector multiplication between the rotation matrix and the vector of the position of the point.
 * Since the position of the points are a double3 type, it is easier to keep track of it.
 * The addition of the PassingPoint (pp) is in order to reposition the point in the space after the first transition needed
 * to compute the right rotation. Each thread address a single atom and the results are stored in block order using the index 
 * variable.
 * 
 * @param res Store the result.
 * @param atoms all the atom to translate.
 * @param number_of_atoms number of the atoms to be rotated each time.
 * @param pp PassingPoint, point belonging to the axis.
 * @param unit_quaternion Array with all the unit quaternions.
 * @param angle Angle of the rotation of the first Block
 **/
__global__ void rotation_kernel_v5(atom_st* res, atom_st* atoms,
                                int number_of_atoms, double3 pp, double4* unit_quaternion, int angle){
    

    int tid = threadIdx.x; 
    if(angle+blockIdx.x < 360){
    
        __shared__ double rot_matrix[3][3];
    
        if(tid == 0) initFirstRow_v2(rot_matrix,unit_quaternion[angle+blockIdx.x]);
        else if(tid == 1) initSecondRow_v2(rot_matrix,unit_quaternion[angle+blockIdx.x]);
        else if(tid == 2) initThirdRow_v2(rot_matrix,unit_quaternion[angle+blockIdx.x]);

    
        __syncthreads();
        // The index variable is needed to compute the right position in the result array, in order
        // not to mix the results between blocks.
        int index = threadIdx.x + blockIdx.x*number_of_atoms;
        if(index < number_of_atoms*(blockIdx.x+1) && number_of_atoms*blockIdx.x <= index){
        
            res[index].id = atoms[tid].id;

            res[index].position.x = atoms[tid].position.x * rot_matrix[0][0] + \
                                atoms[tid].position.y * rot_matrix[0][1] + \
                                atoms[tid].position.z * rot_matrix[0][2] + pp.x;
        
            res[index].position.y = atoms[tid].position.x * rot_matrix[1][0] + \
                                atoms[tid].position.y * rot_matrix[1][1] + \
                                atoms[tid].position.z * rot_matrix[1][2] + pp.y;
            
            res[index].position.z = atoms[tid].position.x * rot_matrix[2][0] + \
                                atoms[tid].position.y * rot_matrix[2][1] + \
                                atoms[tid].position.z * rot_matrix[2][2] + pp.z;
        }
        
    }

}


__global__ void first_translation(atom_st* atoms,double3 pp,int number_of_atoms){

    int tidx = threadIdx.x;
     
    if(tidx < number_of_atoms){
        atoms[tidx].position.x -= pp.x;
        atoms[tidx].position.y -= pp.y;
        atoms[tidx].position.z -= pp.z;
    }

}

__global__ void back_translation(atom_st* atoms,double3 pp,int number_of_atoms){

    int tidx = threadIdx.x;
     
    if(tidx < number_of_atoms){
        atoms[tidx].position.x += pp.x;
        atoms[tidx].position.y += pp.y;
        atoms[tidx].position.z += pp.z;
    }

}


/**
 * This function is used to set the mememory of the host and the device in order to compute the rotation using the rotation kernel.
 * At the end of the computation all the rotated positions are brought to the device memory and stored in a vector of vectors for 
 * future usage.
 * 
 * @param angle The angle of the first rotation of the block.
 * @param atoms_st Vector containing all the atoms to rotate.
 * @param pp PassingPoint, point belonging to the axis of the rotation, used to compute the rotation.
 * @param unit_quaternion The vector containing all the computed unit_quaternions, one for each rotation.
 **/
vector<vector<atom_st>> Rotation::rotate_v5(int angle, std::vector<atom_st>& atoms_st, double3& pp, double4* unit_quaternion){

    int deviceId;
    int number_of_atoms = atoms_st.size();
    int size_of_atoms = number_of_atoms*sizeof(atom_st);

    atom_st *atoms;    
    hipError_t err;

    atom_st * h_res;
    atom_st * d_res;    


    checkCuda( hipGetDevice(&deviceId), __LINE__ );
    hipMallocManaged(&atoms, size_of_atoms);
    
    checkCuda( hipHostMalloc(&h_res, size_of_atoms*NUM_OF_BLOCKS),__LINE__);
    
    checkCuda( hipMalloc(&d_res,size_of_atoms*NUM_OF_BLOCKS), __LINE__);

    
    //initialize vector of atoms
    int i = 0;
    for(auto at : atoms_st){
        atoms[i] = at;
        i++;
    }
    
    checkCuda( hipMemPrefetchAsync(atoms,size_of_atoms, deviceId), __LINE__);
    
    double3 passingPoint = pp;
    
    first_translation<<<1,number_of_atoms>>>(atoms,passingPoint, number_of_atoms);

    checkCuda( hipDeviceSynchronize(), __LINE__);
    checkCuda( hipMemPrefetchAsync(unit_quaternion,360*sizeof(double4), deviceId) ,__LINE__);
    checkCuda( hipMemPrefetchAsync(atoms,size_of_atoms,deviceId), __LINE__);
    
    rotation_kernel_v5<<<NUM_OF_BLOCKS,64,0>>>(d_res,atoms,number_of_atoms,passingPoint,unit_quaternion,angle);

    err = hipGetLastError();
    if(err != hipSuccess){
        cout << __LINE__ << endl;
        printf("Error %s \n", hipGetErrorString(err));
    }    

    checkCuda( hipDeviceSynchronize(),__LINE__);
    
    checkCuda( hipMemcpy(h_res, d_res, size_of_atoms * NUM_OF_BLOCKS, hipMemcpyDeviceToHost), __LINE__ );
    
    checkCuda( hipFree(atoms), __LINE__ );
    
    vector<vector<atom_st>> result_to_return;
    vector<atom_st> tmp;
    //copy the results in order to free the memory and to pass the result to other functions for further usage
    for(int i = 0; i < NUM_OF_BLOCKS; i++ ){
        for(int c = atoms_st.size()*i; c < atoms_st.size()*(i+1); c++){
            tmp.push_back(h_res[c]);
        }
        result_to_return.push_back(tmp);
        tmp.clear();
    }

    checkCuda( hipHostFree(h_res), __LINE__);
    checkCuda( hipFree(d_res),__LINE__ );


    return result_to_return;
}
