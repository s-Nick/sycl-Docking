#include "hip/hip_runtime.h"

#include <GraphMol/GraphMol.h>
#include <GraphMol/FileParsers/MolSupplier.h>
#include <GraphMol/FileParsers/MolWriters.h>
#include <GraphMol/FileParsers/FileParsers.h>

#include <GraphMol/RDKitBase.h>
#include <GraphMol/SmilesParse/SmilesWrite.h>
#include <RDGeneral/FileParseException.h>
#include <RDGeneral/BadFileException.h>

#include <iostream>
#include <algorithm>
#include <vector>
#include <string>
#include <stdio.h>
#include <chrono>

#include "hip/hip_math_constants.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"

#include "helper.h"

#define NUM_OF_BLOCKS 360

using namespace RDKit;

/**
 * Struct used to keep track of the max result found.
 * It keeps track of the distance, the angle and the rotamer. From old version and for future expansions,
 * it keeps also track of the rotated positions of the first half of the molecule. 
 **/
struct max_value{
    double distance;
    int angle;
    Rotamer rt;
    //atom_st* rot_mol_fst_half;
    std::string mol_name;
};


/**
 * Compute the unit quaternion used in the computation of the rotation matrix.
 * Each thread compute one unit_quaternion.
 * 
 * @param res Array with the result.
 * @param quaternion Array containing the data of the vector, which the atoms must rotate around.
 **/
__global__ void compute_unit_quaternions(double4* res, double3 quaternion){

    int tid = threadIdx.x;// + blockIdx.x*gridDim.x;

    double x , y ,z;
    double angle;
    double sin_2 , cos_2;

    //compute the norm of the vector.
    double norm = norm3d(quaternion.x, quaternion.y,quaternion.z);
    if(tid < 360){
        x = quaternion.x/norm;
        y = quaternion.y/norm;
        z = quaternion.z/norm;
        angle = HIP_PI/180 * tid;
        sin_2 = sin(angle/2);
        cos_2 = cos(angle/2);
        res[tid] = make_double4(x*sin_2, y*sin_2 , z*sin_2 , cos_2);//computed accordingly to quaternion explained in the report.
    }

}

void analyzeMolecule(max_value& max_dist, std::shared_ptr<RDKit::ROMol> mol){
    std::vector<Rotamer> rotamers;
    std::vector<atom_st> atoms;

    // Initialize the graph.
    Graph graph = Graph(mol->getNumAtoms());
    
    auto conf = mol->getConformer();
    
    std::cout << "number of bonds: " << mol->getNumBonds() << '\n';// mol2->getNumBonds() << '\n';

    if( !mol->getRingInfo()->isInitialized() ) {
        RDKit::MolOps::findSSSR( *mol );
    }

    //for( unsigned int i = 0; i < mol->getNumBonds() ; i++ ) {
    //    const RDKit::Bond *bond = mol->getBondWithIdx( i );
    //}

    // Get all the Bond in the mol and add the valid ones to the rotamers' vector.
    // Since the Bond in rings and the Double bond are not considerated useful for
    // the rotation, it discards them.
    auto conv_to_double3 = [](const RDGeom::Point3D& pos) {
        return make_double3(pos[0], pos[1], pos[2]);
    };

    for( unsigned int i = 0; i < mol->getNumBonds() ; i++ ) {
        const RDKit::Bond *bond = mol->getBondWithIdx( i );
        unsigned int startingAtom, endingAtom;
        startingAtom = bond->getBeginAtomIdx();
        endingAtom = bond->getEndAtomIdx();
        graph.addEdge(startingAtom,endingAtom);
        if( mol->getRingInfo()->numBondRings( bond->getIdx() )) {
            //continue;
            std::cout <<  "Bond " << bond->getIdx() << " is in a ring " << "stAtom: " << startingAtom << " endAtom: " << endingAtom << "\n";
        }
        else if(bond->getBondType() == RDKit::Bond::BondType::DOUBLE){
            //continue;
            std::cout <<  "Bond " << bond->getIdx() << " is a DOUBLE bond " << "stAtom: " << startingAtom << " endAtom: " << endingAtom << "\n";
        }
        else{
            unsigned int id = bond->getIdx();
            atom_st beginAtom{startingAtom, conv_to_double3(conf.getAtomPos(startingAtom))} ;
            atom_st endAtom{endingAtom, conv_to_double3(conf.getAtomPos(endingAtom))};
            rotamers.push_back(Rotamer(*bond,id, beginAtom, endAtom));
        }
    }


    // Add all the atoms to the atoms' vector
    for(auto atom : mol->atoms()){
        const uint id = atom->getIdx();
        atoms.push_back(atom_st{id,conv_to_double3(conf.getAtomPos(id))});
    }

    //Initialize the result storing structure.
    //max_value max_dist;
    //max_dist.distance = 0;


    max_value max_first_half;
    max_first_half.distance = 0;
    max_value max_second_half;
    max_second_half.distance = 0;

    std::vector<unsigned int> first_half;
    std::vector<unsigned int> second_half;
    //Rotamer rt = rotamers[0];
    //vector<Rotamer> tmp_rotamers ={rotamers[0], rotamers[1]};
    auto start = std::chrono::high_resolution_clock::now();
    // Cycle through all the available rotamers 
    for(auto rt : rotamers){

        bool analize;

        // Removing the analize edge/bond
        graph.removeEdge(rt.getBeginAtom().id, rt.getEndingAtom().id);

        // Compute the two halves of the splitted molecule.
        graph.DFSlinkedNode(rt.getBeginAtom().id, first_half);
        graph.DFSlinkedNode(rt.getEndingAtom().id, second_half);

        std::vector<atom_st> atoms_first_half;
        std::vector<atom_st> atoms_second_half;
        
        for(auto i: first_half)  atoms_first_half.push_back(atoms[i]);
        
        for(auto i : second_half) atoms_second_half.push_back(atoms[i]);
        /*
        max_value max_first_half;
        max_first_half.distance = 0;
        max_value max_second_half;
        max_second_half.distance = 0;
        */

        Rotation r;

        // If the bond split, create one half with only one atom. The bond is not a rotamer,
        // so I don't rotate around it and skip the computation.
        if(atoms_first_half.size() > 1 && second_half.size() > 1){
            
            analize = true;
            std::cout << "Checking rotamer: " << rt.getBond().getIdx() << " ";
            std::cout << "Starting Atom: " << rt.getBeginAtom().id << " Ending Atom: " << rt.getEndingAtom().id << " ";

            std::cout << "number of atom in first half: " << atoms_first_half.size() << "\n";

            std::vector<atom_st> distance_to_compute;
            double4* unit_quaternions;

            hipMallocManaged(&unit_quaternions, 360*sizeof(double4));

            int deviceId;
            hipGetDevice(&deviceId);

            hipMemPrefetchAsync(unit_quaternions,360*sizeof(double4),deviceId);
            
            //Vector of the rotamer considered in the loop.
            double3 tmp_vector = rt.getVector();

            // The computatioin of the unit quaternion is done in parallel for all
            // the angle, launching the kernel with 360 threads, one for each angle.
            compute_unit_quaternions<<<1,360>>>(unit_quaternions,tmp_vector);

            hipDeviceSynchronize();
            
            
            double max = 0;
            double* res = nullptr;
            
            for(int c = 0; c < 360; c += NUM_OF_BLOCKS ){
                
                std::vector<std::vector<atom_st>> rot_first_half;
                
                double3 tmp = rt.getBeginAtom().position;

                // Compute the rotation and storing the result
                rot_first_half = r.rotate_v5(c , atoms_first_half, tmp, unit_quaternions);

                // Add all the element of the vector of vectors in a single vector with all the atoms.
                // The atoms are in order of angle of rotation and every time is added the missing atoms
                // of the second half of the molecule, in order to compute the internal distance.
                for(int rotation = 0; rotation < NUM_OF_BLOCKS; ++rotation){
                    //cout << "main line " << __LINE__ << endl;
                    for(int i = 0; i < atoms_first_half.size(); i++){
                        distance_to_compute.push_back(rot_first_half[rotation][i]);
                    }
                    //cout << "main line " << __LINE__ << endl;
                    for(atom_st at : atoms_second_half){
                        distance_to_compute.push_back(at);
                    }
                }

                // Compute the internal distance, storing the result in res.
                res = distance_v3(distance_to_compute, atoms.size(), NUM_OF_BLOCKS);
                
                // Select the rotation that has the highest internal distance,
                // cycling through the results stored in res. 
                for(int i = 0; i < NUM_OF_BLOCKS; ++i) {
                    if(res[i] > max_first_half.distance) {
                        max_first_half.distance = res[i];
                        max_first_half.angle = c+i;
                        max_first_half.rt = rt;
                    }
                }
                distance_to_compute.clear();
                std::vector<atom_st>().swap(distance_to_compute);

                rot_first_half.clear();
                std::vector<std::vector<atom_st>>().swap(rot_first_half);

            }


            printf("Computed distance for the first part,\n");
            printf("the max distance compute is %lf with angle %d around rotamer: %d\n", \
                    max_first_half.distance, max_first_half.angle,max_first_half.rt.getBond().getIdx());
            
            hipFree(unit_quaternions);
            hipFree(res);
        }
        else{
            analize = false;
            printf("Checking rotamer %d ... ", rt.getBond().getIdx());
            printf("Too few atoms in the partition, rotamer not analized\n");
        }

        double total = max_first_half.distance + max_second_half.distance;

        if(total > max_dist.distance){
            max_dist.distance = total;
            max_dist.rt = max_first_half.rt;
            max_dist.angle = max_first_half.angle;
            max_dist.mol_name = mol->getProp<std::string>("_Name");
        }

        first_half.clear();
        second_half.clear();
        atoms_first_half.clear();
        atoms_second_half.clear();

        // Adding again the edge corresponding to the bond, before computing another bond/rotamer.
        graph.addEdge(rt.getBeginAtom().id,rt.getEndingAtom().id);
        if(analize)
            printf("For Rotamer %d, the max distance computed is: %lf,\n with a first angle: %d \n",\
                rt.getBond().getIdx(),total,max_first_half.angle);

    }

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start);

    std::cout << "duration time[ms]: " << duration.count() << "\n";


    //printf("For molecule named %s \n", mol->getProp<std::string>("_Name") );
    std::cout << "For molecule named " << mol->getProp<std::string>("_Name") << "\n";

    printf("The maximum distance computed is %lf\n", max_first_half.distance);
    
    printf("Computed with an angle of %d, around the rotamer %d\n",
        max_first_half.angle,max_first_half.rt.getBond().getIdx());
    
    return;
}

/**
 * Main function of the code. It parse the file and retrieve all the necessary data for the computation.
 * It takes as input the mol2 file that describe the molecule.
 **/
int main(int argc, char** argv){

    std::string mol_file = argv[1];
    char* mol_number_string = argv[2];
    //std::vector<Rotamer> rotamers;
    //std::vector<atom_st> atoms;
    //RWMol *m = Mol2FileToMol( mol_file );
    //std::shared_ptr<RDKit::ROMol>const  mol( RDKit::Mol2FileToMol( mol_file,true,false,CORINA,false ) );

    /**
     * The following initialization works with the aspirin's mol2 file provided by the Professor.
     * The declaration above works only with the file found online.
     */
    //std::shared_ptr<RDKit::ROMol>const  mol( RDKit::Mol2FileToMol( mol_file,false,true,CORINA,false ) );
    /**The next Line read the molecule removing the H atoms, it reduce the number of possible rotors
     *  for the aspirin and it seems to work, but idk with others molecules, so for now I keep
     * more rotores, but with the possible right solution.
     */
    //std::shared_ptr<RDKit::ROMol> mol( RDKit::Mol2FileToMol( mol_file,true,true,CORINA,false ) );

    
    std::ifstream molFileStream;
    molFileStream.open(mol_file, std::ios::in);

    std::vector<std::shared_ptr<RDKit::ROMol>> molecules;
    //readMoleculesStream(molFileStream, molecules);
    int mol_number = atoi(mol_number_string);
    if(mol_number == 1){
        singleMoleculeRead(molFileStream, molecules);
    }
    else{
        multipleMoleculeRead(molFileStream,molecules);
    }
    //auto tmp = molecules[3]->getProp<std::string>("_Name");
    
    //Initialize the result storing structure.
    max_value max_dist;
    max_dist.distance = 0;


    auto total_start = std::chrono::high_resolution_clock::now();

    std::cout << molecules.size() << "\n";
    
    
    for(auto mol : molecules){
        analyzeMolecule(max_dist, mol);
    }
    
    
    auto final_stop = std::chrono::high_resolution_clock::now();
    auto total_duration = std::chrono::duration_cast<std::chrono::milliseconds>(final_stop-total_start);

    std::cout << "total duration of the computation[ms]: " << total_duration.count() << "\n"; 

    printf("The overall maximum distance computed is %lf ", max_dist.distance);
    std::cout  << " obtained from molecule " <<  max_dist.mol_name << "\n"; 
        
    printf("Computed with an angle of %d, around the rotamer %d\n",max_dist.angle,max_dist.rt.getBond().getIdx());
    
    return 0;
}